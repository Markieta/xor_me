#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>


inline void lclRotateRight(unsigned short& rnValue, size_t nBits)
{
	rnValue = (rnValue >> nBits) | (rnValue << (sizeof(unsigned short) * 8 - nBits));
}

inline void lclRotateLeft(unsigned short& rnValue, size_t nBits)
{
	rnValue = (rnValue << nBits) | (rnValue >> (sizeof(unsigned short) * 8 - nBits));
}

inline signed long lclGetLen(const unsigned char* pnPassData, signed long nBufferSize)
{
	signed long nLen = 0;
	while ( (nLen < nBufferSize) && pnPassData[ nLen ] ) { ++nLen; }
	return nLen;
}

unsigned short lclGetKey(const unsigned char* pnPassData, signed long nBufferSize)
{
	signed long nLen = lclGetLen(pnPassData, nBufferSize);
	if ( nLen <= 0 ) return 0;

	unsigned short nKey = 0;
	unsigned short nKeyBase = 0x8000;
	unsigned short nKeyEnd = 0xFFFF;
	const unsigned char* pnChar = pnPassData + nLen - 1;
	for (signed long nIndex = 0; nIndex < nLen; ++nIndex, --pnChar) {
		unsigned char cChar = *pnChar & 0x7F;
		for (size_t nBit = 0; nBit < 8; ++nBit) {
			lclRotateLeft( nKeyBase, 1 );
			if (nKeyBase & 1) nKeyBase ^= 0x1020;
			if (cChar & 1) nKey ^= nKeyBase;
			cChar >>= 1;
			lclRotateLeft(nKeyEnd, 1);
			if (nKeyEnd & 1) nKeyEnd ^= 0x1020;
		}
	}
	return nKey ^ nKeyEnd;
}

unsigned short lclGetHash(const unsigned char* pnPassData, const unsigned short* pnRotatedData,signed long nBufferSize)
{
	signed long nLen = lclGetLen(pnPassData, nBufferSize);
	unsigned short nHash = static_cast<unsigned short>(nLen) ^ 0xCE4B;

	const unsigned short* pnChar = pnRotatedData;
	for(signed long nIndex = 0; nIndex < nLen; ++nIndex, ++pnChar) {
		nHash ^= *pnChar;
	}
	return nHash;
}

// state for restore in SIGINT handler
unsigned char i, j, k, l, m, n, o;
unsigned short hash;
unsigned short r[9] = {0};
unsigned char t[9] = {1, 0};

#define STATEFORMAT "%02hhx:%02hhx:%02hhx:%02hhx:%02hhx:%02hhx:%02hhx:" \
                    "%04hx:" \
                    "%04hx%04hx%04hx%04hx%04hx%04hx%04hx%04hx:" \
                    "%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx"

#define LOOPSIZE 96
#define RT_SIZE 9

void dump_exit(int) {
	printf("State: " STATEFORMAT "\n",
	       i, j, k, l, m, n, o,
	       hash,
	       r[0], r[1], r[2], r[3], r[4], r[5], r[6], r[7],
	       t[0], t[1], t[2], t[3], t[4], t[5], t[6], t[7]);
	exit(0);
}

void usage_exit(char *prog) {
	std::cout << "Usage: " << prog << " <0xKey> <0xHash>" << std::endl;
	exit(1);
}

__global__ void initialize()
{
	;	
}

int main(int argc, char ** argv) {
	bool state = false;
	char *prog = argv[0];

	if (argc < 2)
		usage_exit(prog);
	if (!strcmp("-s", argv[1])) {
		sscanf(argv[2], STATEFORMAT,
		       &i, &j, &k, &l, &m, &n, &o,
		       &hash,
		       r, r+1, r+2, r+3, r+4, r+5, r+6, r+7,
		       t, t+1, t+2, t+3, t+4, t+5, t+6, t+7);
		state = true;
		argv += 2; argc -= 2;
	}
	if (argc < 3)
		usage_exit(prog);

	signal(SIGINT, &dump_exit);

	unsigned short nKey;
	std::istringstream issk(argv[1]);
	issk >> std::hex >> nKey;
	unsigned short nHash;
	std::istringstream issh(argv[2]);
	issh >> std::hex >> nHash;

	std::cout << std::hex << "Key: " << nKey << std::endl;
	std::cout << std::hex << "Hash: " << nHash << std::endl;

	// unsigned short* h_r = new unsigned short[LOOPSIZE * RT_SIZE];
	// unsigned char*  h_t = new unsigned char[LOOPSIZE  * RT_SIZE];
	// unsigned char*  h_x = new unsigned char[LOOPSIZE];

	thrust::host_vector<unsigned short> h_r(LOOPSIZE * RT_SIZE);
	thrust::host_vector<unsigned short> h_hash(LOOPSIZE);
	thrust::host_vector<unsigned char>  h_t(LOOPSIZE * RT_SIZE);
	thrust::host_vector<unsigned char>  h_x(LOOPSIZE);

	// unsigned short* d_r;
	// unsigned char*  d_t;
	// unsigned char*  d_x;

	// hipMalloc((void**)&d_r, LOOPSIZE * RT_SIZE);
	// hipMalloc((void**)&d_t, LOOPSIZE * RT_SIZE);
	// hipMalloc((void**)&d_x, LOOPSIZE);

	if (state)
		goto skipInits;

	hash = lclGetHash(t, r, 16);
	// BRUTE FORCE up to 8 chars
	for (i=32; i < 127; ++i) {
		for (j=32; j < 128; ++j) {
			for (k=32; k < 128; ++k) {
				for (l=32; l < 128; ++l) {
					for (m=32; m < 128; ++m) {
						for (n=32; n < 128; ++n) {
skipInits:

							// Initial case
							unsigned short x = nHash ^ hash;
							lclRotateRight(x, 1);
							if (32 <= x && x < 127) {
								t[0] = static_cast<unsigned char>(x);
								if (nKey == lclGetKey(t, 16)) {
									std::cout << "Password: '" << t << "'" << std::endl;
								}
							}
							hash ^= r[1];
							r[1] = t[1] = o;
							lclRotateLeft(r[1], 2);
							hash ^= r[1]; 
							// if o == 32
							r[0] = '\0';
							hash = lclGetHash(t, r, 16);

							thrust::device_vector<unsigned short> d_r1(LOOPSIZE);

							int p;

							// Copy in the array t and r, LOOPSIZE times into the host array
							for(p=0; p < LOOPSIZE * RT_SIZE; p++)
							{
								int pos = p % RT_SIZE;
								h_r[p]  = r[pos];
								h_t[p]  = t[pos];
							}

							// Assign relative o to each instance of t[1] and r[1];
							for(o=32, p=0; o < 128; ++o, p++)
							{
								int pos  = p * RT_SIZE + 1;
								h_r[pos] = o;
								h_t[pos] = o;
								d_r1[p]  = o; // For hash ^ r[1] operations
							}

							// hipMemcpy(d_r, h_r, LOOPSIZE * RT_SIZE, hipMemcpyHostToDevice);
							// hipMemcpy(d_t, h_t, LOOPSIZE * RT_SIZE, hipMemcpyHostToDevice);
							// hipMemcpy(d_x, h_x, LOOPSIZE,           hipMemcpyHostToDevice);

							thrust::device_vector<unsigned short> d_r    = h_r;
							thrust::device_vector<unsigned short> d_hash = h_hash;
							thrust::device_vector<unsigned char> d_t     = h_t;
							thrust::device_vector<unsigned char> d_x     = h_x;
							thrust::device_vector<unsigned short> xor_hash(LOOPSIZE); // XOR results

							// For XOR operations with r[1]
							// thrust::device_ptr<unsigned short>       pHash = &d_hash[0];
							// thrust::device_reference<unsigned short> rHash(pHash);
							thrust::transform(d_hash.begin(), d_hash.end(), d_r1.begin(),
									  xor_hash.begin(), thrust::bit_xor<int>());

							/*for (o=32; o < 128; ++o) {
skipInits:
								unsigned short x = nHash ^ hash;
								lclRotateRight(x, 1);
								if (32 <= x && x < 127) {
									t[0] = static_cast<unsigned char>(x);
									if (nKey == lclGetKey(t, 16)) {
										std::cout << "Password: '" << t << "'" << std::endl;
									}
								}
								hash ^= r[1];
								r[1] = t[1] = o;
								lclRotateLeft(r[1], 2);
								hash ^= r[1];
								if (o == 32) {
									r[0] = '\0';
									hash = lclGetHash(t, r, 16);
								}
							}*/
							hash ^= r[2];
							r[2] = t[2] = n;
							lclRotateLeft(r[2], 3);
							hash ^= r[2];
							if (n == 32) {
								r[0] = '\0';
								hash = lclGetHash(t, r, 16);
							}
						}
						hash ^= r[3];
						r[3] = t[3] = m;
						lclRotateLeft(r[3], 4);
						hash ^= r[3];
						if (m == 32) {
							r[0] = '\0';
							hash = lclGetHash(t, r, 16);
						}
					}
					hash ^= r[4];
					r[4] = t[4] = l;
					lclRotateLeft(r[4], 5);
					hash ^= r[4];
					if (l == 32) {
						r[0] = '\0';
						hash = lclGetHash(t, r, 16);
					}
				}
				hash ^= r[5];
				r[5] = t[5] = k;
				lclRotateLeft(r[5], 6);
				hash ^= r[5];
				if (k == 32) {
					r[0] = '\0';
					hash = lclGetHash(t, r, 16);
				}
			}
			hash ^= r[6];
			r[6] = t[6] = j;
			lclRotateLeft(r[6], 7);
			hash ^= r[6];
			if (j == 32) {
				r[0] = '\0';
				hash = lclGetHash(t, r, 16);
			}
		}
		hash ^= r[7];
		r[7] = t[7] = i;
		lclRotateLeft(r[7], 8);
		hash ^= r[7];
		if (i == 32) {
			r[0] = '\0';
			hash = lclGetHash(t, r, 16);
		}
	}
}
